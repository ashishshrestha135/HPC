#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
/***********************************************************************
**

  Compile with:
    nvcc -o CrackAZ99-With-Data-cuda CrackAZ99-With-Data-cuda.cu 

    ./CrackAZ99-With-Data-cuda

  Dr Kevan Buckley, University of Wolverhampton, 2018
************************************************************************
******/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
  char plain_password[] = "ML42";
  
  char *a = attempt;
  char *p = plain_password;
  
  while(*a == *p) {
    if(*a == '\0') {
      return 1;
    }
    a++;
    p++;
  }
  return 0;
}

/****************************************************************************
  The kernel function run in 675 threads uses 
  nested loops to generate all possible passwords and test whether they match
  the hidden password.
*****************************************************************************/

__global__ void kernel() {
  char alpha[26]= {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
 
  char num[10] = {'0','1','2','3','4','5','6','7','8','9'};
  
  char result[5];
  result[4] = '\0';
  int e, f;
  for(e=0;e<=9;e++) {
     for(f=0; f<9; f++) {
          result[0] = alpha[blockIdx.x];
          result[1] = alpha[threadIdx.x];
          result[2] = num[e];
          result[3] = num[f];
          if(is_a_match(result)) {
          printf("password found: %s\n", result);
          } else {
               //printf("tried: %s\n", result);
          }
     }
  }
}


 int time_difference(struct timespec *start, struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}

int main(int argc, char *argv[]) {
  struct timespec start, finish
;   
  long long int time_elapsed;

  clock_gettime(CLOCK_MONOTONIC, &start);
 
  
     kernel <<<26, 26>>>();
  hipDeviceSynchronize();
    
    
  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
                                         (time_elapsed/1.0e9)); 


  return 0;
}
